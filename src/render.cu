#include "hip/hip_runtime.h"
#include "maths.h"
#include "render.h"
#include "util.h"
#include "disney.h"

#include <map>

struct GPUScene
{
	Primitive* primitives;
	int numPrimitives;

	Primitive* lights;
	int numLights;

	Sky sky;
};

#define kBrdfSamples 1.0f
#define kProbeSamples 1.0f
#define kRayEpsilon 0.001f

MeshGeometry CreateGPUMesh(const MeshGeometry& hostMesh)
{
	const int numVertices = hostMesh.numVertices;
	const int numIndices = hostMesh.numIndices;
	const int numNodes = hostMesh.numNodes;

	MeshGeometry gpuMesh;
	hipMalloc(&gpuMesh.positions, sizeof(Vec3)*numVertices);
	hipMemcpy((Vec3*)gpuMesh.positions, &hostMesh.positions[0], sizeof(Vec3)*numVertices, hipMemcpyHostToDevice);

	hipMalloc(&gpuMesh.normals, sizeof(Vec3)*numVertices);
	hipMemcpy((Vec3*)gpuMesh.normals, &hostMesh.normals[0], sizeof(Vec3)*numVertices, hipMemcpyHostToDevice);

	hipMalloc(&gpuMesh.indices, sizeof(int)*numIndices);
	hipMemcpy((int*)gpuMesh.indices, &hostMesh.indices[0], sizeof(int)*numIndices, hipMemcpyHostToDevice);

	hipMalloc(&gpuMesh.nodes, sizeof(BVHNode)*numNodes);
	hipMemcpy((BVHNode*)gpuMesh.nodes, &hostMesh.nodes[0], sizeof(BVHNode)*numNodes, hipMemcpyHostToDevice);
	
	gpuMesh.numIndices = numIndices;
	gpuMesh.numVertices = numVertices;
	gpuMesh.numNodes = numNodes;

	return gpuMesh;

}

void DestroyGPUMesh(const MeshGeometry& m)
{

}

Sky CreateGPUSky(const Sky& sky)
{
	Sky gpuSky = sky;

	// copy probe
	if (sky.probe.valid)
	{
		const int numPixels = sky.probe.width*sky.probe.height;

		// copy pixel data
		hipMalloc(&gpuSky.probe.data, numPixels*sizeof(float)*4);
		hipMemcpy(gpuSky.probe.data, sky.probe.data, numPixels*sizeof(float)*4, hipMemcpyHostToDevice);

		// copy cdf tables
		hipMalloc(&gpuSky.probe.cdfValuesX, numPixels*sizeof(float));
		hipMemcpy(gpuSky.probe.cdfValuesX, sky.probe.cdfValuesX, numPixels*sizeof(float), hipMemcpyHostToDevice);

		hipMalloc(&gpuSky.probe.cdfValuesY, sky.probe.height*sizeof(float));
		hipMemcpy(gpuSky.probe.cdfValuesY, sky.probe.cdfValuesY, sky.probe.height*sizeof(float), hipMemcpyHostToDevice);

		// copy pdf tables
		hipMalloc(&gpuSky.probe.pdfValuesX, numPixels*sizeof(float));
		hipMemcpy(gpuSky.probe.pdfValuesX, sky.probe.pdfValuesX, numPixels*sizeof(float), hipMemcpyHostToDevice);

		hipMalloc(&gpuSky.probe.pdfValuesY, sky.probe.height*sizeof(float));
		hipMemcpy(gpuSky.probe.pdfValuesY, sky.probe.pdfValuesY, sky.probe.height*sizeof(float), hipMemcpyHostToDevice);

	}

	return gpuSky;
}

void DestroyGPUSky(const Sky& gpuSky)
{
	if (gpuSky.probe.valid)
	{
		hipFree(gpuSky.probe.data);
	}
}


// trace a ray against the scene returning the closest intersection
__device__ bool Trace(const GPUScene& scene, const Ray& ray, float& outT, Vec3& outNormal, const Primitive** outPrimitive)
{
	// disgard hits closer than this distance to avoid self intersection artifacts
	float minT = REAL_MAX;
	const Primitive* closestPrimitive = NULL;
	Vec3 closestNormal(0.0f);

	for (int i=0; i < scene.numPrimitives; ++i)
	{
		const Primitive& primitive = scene.primitives[i];

		float t;
		Vec3 n;

		if (Intersect(primitive, ray, t, &n))
		{
			if (t < minT && t > 0.0f)
			{
				minT = t;
				closestPrimitive = &primitive;
				closestNormal = n;
			}
		}
	}
	
	outT = minT;		
	outNormal = closestNormal;
	*outPrimitive = closestPrimitive;

	return closestPrimitive != NULL;
}


__device__ inline Color SampleLights(const GPUScene& scene, const Primitive& surfacePrimitive, const Vec3& surfacePos, const Vec3& surfaceNormal, const Vec3& wo, float time, Random& rand)
{	
	Color sum(0.0f);

	if (scene.sky.probe.valid)
	{
		for (int i=0; i < kProbeSamples; ++i)
		{

			Color skyColor;
			float skyPdf;
			Vec3 wi;

			ProbeSample(scene.sky.probe, wi, skyColor, skyPdf, rand);
			
			/*
			wi = UniformSampleSphere(rand);
			skyColor = ProbeEval(scene.sky.probe, ProbeDirToUV(wi));
			skyPdf = 0.5f*kInv2Pi;
			*/	
			
			
			if (Dot(wi, surfaceNormal) <= 0.0f)
				continue;

			// check if occluded
			float t;
			Vec3 n;
			const Primitive* hit;
			if (Trace(scene, Ray(surfacePos, wi, time), t, n, &hit) == false)
			{
				float brdfPdf = BRDFPdf(surfacePrimitive.material, surfacePos, surfaceNormal, wo, wi);
				Color f = BRDFEval(surfacePrimitive.material, surfacePos, surfaceNormal, wo, wi);
				
				int N = kProbeSamples+kBrdfSamples;
				float cbrdf = kBrdfSamples/N;
				float csky = float(kProbeSamples)/N;
				float weight = csky*skyPdf/(cbrdf*brdfPdf + csky*skyPdf);

				Validate(weight);

				if (weight > 0.0f)
					sum += weight*skyColor*f*Abs(Dot(wi, surfaceNormal))/skyPdf;
			}
		}

		if (kProbeSamples > 0)
			sum /= float(kProbeSamples);
	}

	for (int i=0; i < scene.numLights; ++i)
	{
		// assume all lights are area lights for now
		const Primitive& lightPrimitive = scene.lights[i];

		Color L(0.0f);

		int numSamples = lightPrimitive.lightSamples;

		if (numSamples == 0)
			continue;

		for (int s=0; s < numSamples; ++s)
		{
			// sample light source
			Vec3 lightPos;
			Vec3 lightNormal;

			LightSample(lightPrimitive, time, lightPos, lightNormal, rand);
			
			Vec3 wi = lightPos-surfacePos;
			
			float dSq = LengthSq(wi);
			wi /= sqrtf(dSq);

			// light is behind surface
			if (Dot(wi, surfaceNormal) <= 0.0f)
				continue; 				

			// surface is behind light
			if (Dot(wi, lightNormal) >= 0.0f)
				continue;

			// check visibility
			float t;
			Vec3 n;
			const Primitive* hit;
			if (Trace(scene, Ray(surfacePos, wi, time), t, n, &hit))			
			{
				float tSq = t*t;

				// if our next hit was further than distance to light then accept
				// sample, this works for portal sampling where you have a large light
				// that you sample through a small window
				const float kTolerance = 1.e-2f;

				if (fabsf(t - sqrtf(dSq)) <= kTolerance)
				{				
					const float nl = Dot(lightNormal, -wi);

					// light pdf with respect to area and convert to pdf with respect to solid angle
					float lightArea = LightArea(lightPrimitive);
					float lightPdf = ((1.0f/lightArea)*tSq)/nl;

					// brdf pdf for light's direction
					float brdfPdf = BRDFPdf(surfacePrimitive.material, surfacePos, surfaceNormal, wo, wi);
					Color f = BRDFEval(surfacePrimitive.material, surfacePos, surfaceNormal, wo, wi);

					// calculate relative weighting of the light and brdf sampling
					int N = lightPrimitive.lightSamples+kBrdfSamples;
					float cbrdf = kBrdfSamples/N;
					float clight = float(lightPrimitive.lightSamples)/N;
					float weight = clight*lightPdf/(cbrdf*brdfPdf + clight*lightPdf);
						
					L += weight*f*hit->material.emission*(Abs(Dot(wi, surfaceNormal))/Max(1.e-3f, lightPdf));
				}
			}
		}
	
		sum += L * (1.0f/numSamples);
	}

	return sum;
}


// reference, no light sampling, uniform hemisphere sampling
__device__ Color PathTrace(const GPUScene& scene, const Vec3& origin, const Vec3& dir, float time, int maxDepth, Random& rand)
{	
    // path throughput
    Color pathThroughput(1.0f, 1.0f, 1.0f, 1.0f);
    // accumulated radiance
    Color totalRadiance(0.0f);

	Vec3 rayOrigin = origin;
	Vec3 rayDir = dir;
	float rayTime = time;

    float t = 0.0f;
    Vec3 n(rayDir);
    const Primitive* hit;

	float brdfPdf = 1.0f;

    for (int i=0; i < maxDepth; ++i)
    {
        // find closest hit
        if (Trace(scene, Ray(rayOrigin, rayDir, rayTime), t, n, &hit))
        {	
#if 1
			
			if (i == 0)
			{
				// first trace is our only chance to add contribution from directly visible light sources        
				totalRadiance += hit->material.emission;
			}			
			else if (kBrdfSamples > 0)
			{
				// area pdf that this dir was already included by the light sampling from previous step
				float lightArea = LightArea(*hit);

				if (lightArea > 0.0f)
				{
					// convert to pdf with respect to solid angle
					float lightPdf = ((1.0f/lightArea)*t*t)/Clamp(Dot(-rayDir, n), 1.e-3f, 1.0f);

					// calculate weight for brdf sampling
					int N = hit->lightSamples+kBrdfSamples;
					float cbrdf = kBrdfSamples/N;
					float clight = float(hit->lightSamples)/N;
					float weight = cbrdf*brdfPdf/(cbrdf*brdfPdf+ clight*lightPdf);
							
					Validate(weight);

					// pathThroughput already includes the brdf pdf
					totalRadiance += weight*pathThroughput*hit->material.emission;
				}
			}

            // calculate a basis for this hit point
            Vec3 u, v;
            BasisFromVector(n, &u, &v);

            const Vec3 p = rayOrigin + rayDir*t + n*kRayEpsilon;

			// integrate direct light over hemisphere
			totalRadiance += pathThroughput*SampleLights(scene, *hit, p, n, -rayDir, rayTime, rand);
#else
			
			// calculate a basis for this hit point
            Vec3 u, v;
            BasisFromVector(n, &u, &v);

            const Vec3 p = rayOrigin + rayDir*t + n*kRayEpsilon;

			totalRadiance += pathThroughput*hit->material.emission;

#endif

			// integrate indirect light by sampling BRDF
			Mat33 localFrame(u, v, n);

            Vec3 brdfDir = BRDFSample(hit->material, p, Mat33(u, v, n), -rayDir, rand);
			brdfPdf = BRDFPdf(hit->material, p, n, -rayDir, brdfDir);

			
            if (brdfPdf <= 0.0f)
            	break;

            if (Dot(brdfDir, n) <= 0.0f)
            	break;
				

			Validate(brdfPdf);


            // reflectance
            Color f = BRDFEval(hit->material, p, n, -rayDir, brdfDir);

            // update throughput with primitive reflectance
            pathThroughput *= f * Clamp(Dot(n, brdfDir), 0.0f, 1.0f)/brdfPdf;

            // update path direction
            rayDir = brdfDir;
            rayOrigin = p;
        }
        else
        {
            // hit nothing, sample sky dome and terminate         
            float weight = 1.0f;

        	if (scene.sky.probe.valid && i > 0)
        	{ 
        		// probability that this dir was already sampled by probe sampling
        		float skyPdf = ProbePdf(scene.sky.probe, rayDir);
				 
				int N = kProbeSamples+kBrdfSamples;
				float cbrdf = kBrdfSamples/N;
				float csky = float(kProbeSamples)/N;
			
				weight = cbrdf*brdfPdf/(cbrdf*brdfPdf+ csky*skyPdf);

				Validate(brdfPdf);
				Validate(skyPdf);

			}

			Validate(weight);
		
       		totalRadiance += weight*scene.sky.Eval(rayDir)*pathThroughput; 
			break;
        }
    }

    return totalRadiance;
}

__device__ void AddSample(Color* output, int width, int height, float rasterX, float rasterY, float clamp, Filter filter, const Color& sample)
{
	switch (filter.type)
	{
		case eFilterBox:
		{
			int x = int(rasterX);
			int y = int(rasterY);

			output[y*width+x] += Color(sample.x, sample.y, sample.z, 1.0f);
			break;
		}
		case eFilterGaussian:
		{
			int startX = Max(0, int(rasterX - filter.width));
			int startY = Max(0, int(rasterY - filter.width));
			int endX = Min(int(rasterX + filter.width), width-1);
			int endY = Min(int(rasterY + filter.width), height-1);

			Color c =  ClampLength(sample, clamp);
			c.w = 1.0f;

			for (int x=startX; x <= endX; ++x)
			{
				for (int y=startY; y <= endY; ++y)
				{
					float w = filter.Eval(x-rasterX, y-rasterY);

					//output[(height-1-y)*width+x] += Color(Min(sample.x, clamp), Min(sample.y, clamp), Min(sample.z, clamp), 1.0f)*w;

					const int index = y*width+x;

					atomicAdd(&output[index].x, c.x*w);
					atomicAdd(&output[index].y, c.y*w);
					atomicAdd(&output[index].z, c.z*w);
					atomicAdd(&output[index].w, w);
				}
			}
		
			break;
		}
	};
}

__global__ void RenderGpu(GPUScene scene, Camera camera, CameraSampler sampler, Options options, int seed, Color* output)
{
	const int tid = blockDim.x*blockIdx.x + threadIdx.x;

	const int i = tid%options.width;
	const int j = tid/options.width;

	if (i < options.width && j < options.height)
	{
		Vec3 origin;
		Vec3 dir;

		// initialize a per-thread PRNG
		Random rand(tid + seed);

		if (options.mode == eNormals)
		{
			sampler.GenerateRay(i, j, origin, dir);

			const Primitive* p;
			float t;
			Vec3 n;

			if (Trace(scene, Ray(origin, dir, 1.0f), t, n, &p))
			{
				n = n*0.5f+0.5f;
				output[j*options.width+i] = Color(n.x, n.y, n.z, 1.0f);
			}
			else
			{
				output[j*options.width+i] = Color(0.5f);
			}
		}
		else if (options.mode == ePathTrace)
		{
			const float time = rand.Randf(camera.shutterStart, camera.shutterEnd);
			const float fx = i + rand.Randf(-0.5f, 0.5f) + 0.5f;
			const float fy = j + rand.Randf(-0.5f, 0.5f) + 0.5f;

			Vec3 origin, dir;
			sampler.GenerateRay(fx, fy, origin, dir);

			//output[(height-1-j)*width+i] += PathTrace(*scene, origin, dir);
			Color sample = PathTrace(scene, origin, dir, time, options.maxDepth, rand);

			AddSample(output, options.width, options.height, fx, fy, options.clamp, options.filter, sample);
		}
	}
}

struct GpuRenderer : public Renderer
{
	Color* output = NULL;
	
	GPUScene sceneGPU;
	
	Random seed;

	// map id to geometry struct
	std::map<int, MeshGeometry> gpuMeshes;

	GpuRenderer(const Scene* s)
	{
		// build GPU primitive and light lists
		std::vector<Primitive> primitives;		
		std::vector<Primitive> lights;

		for (int i=0; i < s->primitives.size(); ++i)
		{
			Primitive primitive = s->primitives[i];

			if (primitive.lightSamples)
			{
				lights.push_back(primitive);
			}

			// if mesh primitive then copy to the GPU
			if (primitive.type == eMesh)
			{
				// see if we have already uploaded the mesh to the GPU
				if (gpuMeshes.find(primitive.mesh.id) == gpuMeshes.end())
				{
					MeshGeometry geo = CreateGPUMesh(primitive.mesh);
					gpuMeshes[geo.id] = geo;

					// replace CPU mesh with GPU copy
					primitive.mesh = geo;
				}
			}	
			
			primitives.push_back(primitive);
		}

		// upload to the GPU
		sceneGPU.numPrimitives = primitives.size();
		sceneGPU.numLights = lights.size();

		if (sceneGPU.numLights > 0)
		{
			hipMalloc(&sceneGPU.lights, sizeof(Primitive)*lights.size());
			hipMemcpy(sceneGPU.lights, &lights[0], sizeof(Primitive)*lights.size(), hipMemcpyHostToDevice);
		}

		if (sceneGPU.numPrimitives > 0)
		{
			hipMalloc(&sceneGPU.primitives, sizeof(Primitive)*primitives.size());
			hipMemcpy(sceneGPU.primitives, &primitives[0], sizeof(Primitive)*primitives.size(), hipMemcpyHostToDevice);
		}

		// copy sky and probe texture
		sceneGPU.sky = CreateGPUSky(s->sky);
	}

	virtual ~GpuRenderer()
	{
		hipFree(output);
		hipFree(sceneGPU.primitives);
		hipFree(sceneGPU.lights);
	}
	
	void Init(int width, int height)
	{
		hipFree(output);
		hipMalloc(&output, sizeof(Color)*width*height);
		hipMemset(output, 0, sizeof(Color)*width*height);
	}

	void Render(const Camera& camera, const Options& options, Color* outputHost)
	{
		const int numThreads = options.width*options.height;
		const int kNumThreadsPerBlock = 256;
		const int kNumBlocks = (numThreads + kNumThreadsPerBlock - 1) / (kNumThreadsPerBlock);
	
		// create a sampler for the camera
		CameraSampler sampler(
			Transform(camera.position, camera.rotation),
			camera.fov, 
			0.001f,
			1.0f,
			options.width,
			options.height);

		RenderGpu<<<kNumBlocks, kNumThreadsPerBlock>>>(sceneGPU, camera, sampler, options, seed.Rand(), output);

		// copy back to output
		hipMemcpy(outputHost, output, sizeof(Color)*numThreads, hipMemcpyDeviceToHost);
	}
};


Renderer* CreateGpuRenderer(const Scene* s)
{
	return new GpuRenderer(s);
}
