#include "hip/hip_runtime.h"
#include "maths.h"
#include "render.h"
#include "util.h"
#include "disney.h"

#include <map>

struct GPUScene
{
	Primitive* primitives;
	int numPrimitives;

	Primitive* lights;
	int numLights;

	Sky sky;
};

#define kBsdfSamples 1.0f
#define kProbeSamples 1.0f
#define kRayEpsilon 0.0001f

// create a texture object from memory and store it in a 64-bit pointer
void CreateIntTexture(int** deviceBuffer, const int* hostBuffer, int sizeInBytes)
{
	int* buffer;
	hipMalloc(&buffer, sizeInBytes);
	hipMemcpy(buffer, hostBuffer, sizeInBytes, hipMemcpyHostToDevice);

#if USE_TEXTURES

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = (void*)buffer;
	resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
	resDesc.res.linear.desc.x = 32; // bits per channel
	resDesc.res.linear.sizeInBytes = sizeInBytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	// cast to pointer
	*deviceBuffer = (int*)tex;
#else

	*deviceBuffer = buffer;

#endif
}

// create a texture object from memory and store it in a 64-bit pointer
void CreateFloatTexture(float** deviceBuffer, const float* hostBuffer, int sizeInBytes)
{
	float* buffer;
	hipMalloc(&buffer, sizeInBytes);
	hipMemcpy(buffer, hostBuffer, sizeInBytes, hipMemcpyHostToDevice);

#if USE_TEXTURES

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = (void*)buffer;
	resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc.res.linear.desc.x = 32; // bits per channel
	resDesc.res.linear.sizeInBytes = sizeInBytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	// cast to pointer
	*deviceBuffer = (float*)tex;

#else

	*deviceBuffer = buffer;

#endif
}

// create a texture object from memory and store it in a 64-bit pointer
void CreateVec4Texture(Vec4** deviceBuffer, const Vec4* hostBuffer, int sizeInBytes)
{
	Vec4* buffer;
	hipMalloc(&buffer, sizeInBytes);
	hipMemcpy(buffer, hostBuffer, sizeInBytes, hipMemcpyHostToDevice);

#if USE_TEXTURES

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = (void*)buffer;
	resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc.res.linear.desc.x = 32; // bits per channel
	resDesc.res.linear.desc.y = 32; // bits per channel
	resDesc.res.linear.desc.z = 32; // bits per channel
	resDesc.res.linear.desc.w = 32; // bits per channel
	resDesc.res.linear.sizeInBytes = sizeInBytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	// cast to pointer
	*deviceBuffer = (Vec4*)tex;

#else

	*deviceBuffer = buffer;

#endif

}


MeshGeometry CreateGPUMesh(const MeshGeometry& hostMesh)
{
	const int numVertices = hostMesh.numVertices;
	const int numIndices = hostMesh.numIndices;
	const int numNodes = hostMesh.numNodes;

	MeshGeometry gpuMesh;
	CreateFloatTexture((float**)&gpuMesh.positions, (float*)&hostMesh.positions[0], sizeof(Vec3)*numVertices);
	CreateFloatTexture((float**)&gpuMesh.normals, (float*)&hostMesh.normals[0], sizeof(Vec3)*numVertices);
	CreateIntTexture((int**)&gpuMesh.indices, (int*)&hostMesh.indices[0], sizeof(int)*numIndices);
	

	/*
	hipMalloc((Vec3**)&gpuMesh.positions, sizeof(Vec3)*numVertices);
	hipMemcpy((Vec3*)gpuMesh.positions, &hostMesh.positions[0], sizeof(Vec3)*numVertices, hipMemcpyHostToDevice);

	hipMalloc((Vec3**)&gpuMesh.normals, sizeof(Vec3)*numVertices);
	hipMemcpy((Vec3*)gpuMesh.normals, &hostMesh.normals[0], sizeof(Vec3)*numVertices, hipMemcpyHostToDevice);

	hipMalloc((int**)&gpuMesh.indices, sizeof(int)*numIndices);
	hipMemcpy((int*)gpuMesh.indices, &hostMesh.indices[0], sizeof(int)*numIndices, hipMemcpyHostToDevice);
	*/
	//hipMalloc((BVHNode**)&gpuMesh.nodes, sizeof(BVHNode)*numNodes);
	//hipMemcpy((BVHNode*)gpuMesh.nodes, &hostMesh.nodes[0], sizeof(BVHNode)*numNodes, hipMemcpyHostToDevice);

	CreateVec4Texture((Vec4**)&gpuMesh.nodes, (Vec4*)&hostMesh.nodes[0], sizeof(BVHNode)*numNodes);
	
	hipMalloc((float**)&gpuMesh.cdf, sizeof(float)*numIndices/3);
	hipMemcpy((float*)gpuMesh.cdf, &hostMesh.cdf[0], sizeof(float)*numIndices/3, hipMemcpyHostToDevice);
	
	gpuMesh.numIndices = numIndices;
	gpuMesh.numVertices = numVertices;
	gpuMesh.numNodes = numNodes;
	gpuMesh.area = hostMesh.area;

	return gpuMesh;

}

void DestroyGPUMesh(const MeshGeometry& m)
{

}


Texture CreateGPUTexture(const Texture& tex)
{
	const int numTexels = tex.width*tex.height*tex.depth;
	
	Texture gpuTex = tex;

	hipMalloc((void**)&gpuTex.data, sizeof(float)*numTexels);
	hipMemcpy(gpuTex.data, tex.data, sizeof(float)*numTexels, hipMemcpyHostToDevice);

	return gpuTex;
}

Sky CreateGPUSky(const Sky& sky)
{
	Sky gpuSky = sky;

	// copy probe
	if (sky.probe.valid)
	{
		const int numPixels = sky.probe.width*sky.probe.height;

		// copy pixel data
		CreateVec4Texture((Vec4**)&gpuSky.probe.data, sky.probe.data, numPixels*sizeof(float)*4);

		// copy cdf tables
		CreateFloatTexture((float**)&gpuSky.probe.cdfValuesX, sky.probe.cdfValuesX, numPixels*sizeof(float));
		CreateFloatTexture((float**)&gpuSky.probe.pdfValuesX, sky.probe.pdfValuesX, numPixels*sizeof(float));

		CreateFloatTexture((float**)&gpuSky.probe.cdfValuesY, sky.probe.cdfValuesY, sky.probe.height*sizeof(float));
		CreateFloatTexture((float**)&gpuSky.probe.pdfValuesY, sky.probe.pdfValuesY, sky.probe.height*sizeof(float));
	}

	return gpuSky;
}

void DestroyGPUSky(const Sky& gpuSky)
{
	if (gpuSky.probe.valid)
	{
		// todo
	}
}


// trace a ray against the scene returning the closest intersection
__device__ bool Trace(const GPUScene& scene, const Ray& ray, float& outT, Vec3& outNormal, const Primitive** outPrimitive)
{
	float minT = REAL_MAX;
	const Primitive* closestPrimitive = NULL;
	Vec3 closestNormal(0.0f);

	for (int i=0; i < scene.numPrimitives; ++i)
	{
		const Primitive& primitive = scene.primitives[i];

		float t;
		Vec3 n;

		if (Intersect(primitive, ray, t, &n))
		{
			if (t < minT && t > 0.0f)
			{
				minT = t;
				closestPrimitive = &primitive;
				closestNormal = n;
			}
		}
	}
	
	outT = minT;		
	outNormal = FaceForward(closestNormal, -ray.dir);
	*outPrimitive = closestPrimitive;

	return closestPrimitive != NULL;
}

__device__ inline float SampleTexture(const Texture& map, int i, int j, int k)
{
	int x = int(Abs(i))%map.width;
	int y = int(Abs(j))%map.height;
	int z = int(Abs(k))%map.depth;
	
	return map.data[z*map.width*map.height + y*map.width + x];
}


__device__ inline float LinearInterp(const Texture& map, const Vec3& pos) 
{
	int i = floorf(pos.x*map.width);
	int j = floorf(pos.y*map.height);
	int k = floorf(pos.z*map.depth);
		
	// trilinear interpolation
	float tx = pos.x*map.width-i;
	float ty = pos.y*map.height-j;
	float tz = pos.z*map.depth-k;
		
	float a = Lerp(SampleTexture(map, i, j, k), SampleTexture(map, i, j, k+1), tz);
	float b = Lerp(SampleTexture(map, i+1, j, k), SampleTexture(map, i+1, j, k+1), tz);
	float c = Lerp(SampleTexture(map, i, j+1, k), SampleTexture(map, i, j+1, k+1), tz);		
	float d = Lerp(SampleTexture(map, i+1, j+1, k), SampleTexture(map, i+1, j+1, k+1), tz);
		
	float e = Lerp(a, b, tx);
	float f = Lerp(c, d, tx);
		
	float g = Lerp(e, f, ty);
		
	return g;
}

__device__ inline Vec3 EvaluateBumpNormal(const Vec3& surfaceNormal, const Vec3& surfacePos, const Texture& bumpMap, const Vec3& bumpTile, float bumpStrength, Random& rand)
{
	Vec3 u, v;
	BasisFromVector(surfaceNormal, &u, &v);

	float eps = 0.01f;

	Vec3 dpdu = u + bumpStrength*surfaceNormal*(LinearInterp(bumpMap, bumpTile*(surfacePos)+u*eps) - LinearInterp(bumpMap, bumpTile*surfacePos))/eps;
	Vec3 dpdv = v + bumpStrength*surfaceNormal*(LinearInterp(bumpMap, bumpTile*(surfacePos)+v*eps) - LinearInterp(bumpMap, bumpTile*surfacePos))/eps;

	return SafeNormalize(Cross(dpdu, dpdv), surfaceNormal);
}



__device__ inline Color SampleLights(const GPUScene& scene, const Primitive& surfacePrimitive, float etaI, float etaO, const Vec3& surfacePos, const Vec3& surfaceNormal, const Vec3& shadingNormal, const Vec3& wo, float time, Random& rand)
{	
	Color sum(0.0f);

	if (scene.sky.probe.valid)
	{
		for (int i=0; i < kProbeSamples; ++i)
		{

			Color skyColor;
			float skyPdf;
			Vec3 wi;

			ProbeSample(scene.sky.probe, wi, skyColor, skyPdf, rand);
			
			/*
			wi = UniformSampleSphere(rand);
			skyColor = ProbeEval(scene.sky.probe, ProbeDirToUV(wi));
			skyPdf = 0.5f*kInv2Pi;
			*/	
			
			
			//if (Dot(wi, surfaceNormal) <= 0.0f)
//				continue;

			// check if occluded
			float t;
			Vec3 n;
			const Primitive* hit;
			if (Trace(scene, Ray(surfacePos + FaceForward(surfaceNormal, wi)*kRayEpsilon, wi, time), t, n, &hit) == false)
			{
				float bsdfPdf = BSDFPdf(surfacePrimitive.material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
				Color f = BSDFEval(surfacePrimitive.material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
				
				if (bsdfPdf > 0.0f)
				{
					int N = kProbeSamples+kBsdfSamples;
					float cbsdf = kBsdfSamples/N;
					float csky = float(kProbeSamples)/N;
					float weight = csky*skyPdf/(cbsdf*bsdfPdf + csky*skyPdf);

					Validate(weight);

					if (weight > 0.0f)
						sum += weight*skyColor*f*Abs(Dot(wi, surfaceNormal))/skyPdf;
				}
			}
		}

		if (kProbeSamples > 0)
			sum /= float(kProbeSamples);
	}

	for (int i=0; i < scene.numLights; ++i)
	{
		// assume all lights are area lights for now
		const Primitive& lightPrimitive = scene.lights[i];

		Color L(0.0f);

		int numSamples = lightPrimitive.lightSamples;

		if (numSamples == 0)
			continue;

		for (int s=0; s < numSamples; ++s)
		{
			// sample light source
			Vec3 lightPos;
			Vec3 lightNormal;

			LightSample(lightPrimitive, time, lightPos, lightNormal, rand);
			
			Vec3 wi = lightPos-surfacePos;
			
			float dSq = LengthSq(wi);
			wi /= sqrtf(dSq);


			// light is behind surface
			//if (Dot(wi, surfaceNormal) <= 0.0f)
				//continue; 				

			// surface is behind light
			if (Dot(wi, lightNormal) >= 0.0f)
				continue;

			// check visibility
			float t;
			Vec3 n;
			const Primitive* hit;
			if (Trace(scene, Ray(surfacePos + FaceForward(surfaceNormal, wi)*kRayEpsilon, wi, time), t, n, &hit))			
			{
				float tSq = t*t;

				// if our next hit was further than distance to light then accept
				// sample, this works for portal sampling where you have a large light
				// that you sample through a small window
				const float kTolerance = 1.e-2f;

				if (fabsf(t - sqrtf(dSq)) <= kTolerance)
				{				
					const float nl = Abs(Dot(lightNormal, wi));

					// light pdf with respect to area and convert to pdf with respect to solid angle
					float lightArea = LightArea(lightPrimitive);
					float lightPdf = ((1.0f/lightArea)*tSq)/nl;

					// bsdf pdf for light's direction
					float bsdfPdf = BSDFPdf(surfacePrimitive.material, etaI, etaO, surfacePos, shadingNormal, wo, wi);
					Color f = BSDFEval(surfacePrimitive.material, etaI, etaO, surfacePos, shadingNormal, wo, wi);

					// this branch is only necessary to exclude specular paths from light sampling
					// todo: make BSDFEval alwasy return zero for pure specular paths and roll specular eval into BSDFSample()
					if (bsdfPdf > 0.0f)
					{
						// calculate relative weighting of the light and bsdf sampling
						int N = lightPrimitive.lightSamples+kBsdfSamples;
						float cbsdf = kBsdfSamples/N;
						float clight = float(lightPrimitive.lightSamples)/N;
						float weight = clight*lightPdf/(cbsdf*bsdfPdf + clight*lightPdf);
						
						L += weight*f*hit->material.emission*(Abs(Dot(wi, shadingNormal))/Max(1.e-3f, lightPdf));
					}
				}
			}
		}
	
		sum += L * (1.0f/numSamples);
	}

	return sum;
}


// reference, no light sampling, uniform hemisphere sampling
__device__ Color PathTrace(const GPUScene& scene, const Vec3& origin, const Vec3& dir, float time, int maxDepth, Random& rand)
{	
    // path throughput
    Color pathThroughput(1.0f, 1.0f, 1.0f, 1.0f);
    // accumulated radiance
    Color totalRadiance(0.0f);

	Vec3 rayOrigin = origin;
	Vec3 rayDir = dir;
	float rayTime = time;
	float rayEta = 1.0f;
	Vec3 rayAbsorption = 0.0f;
	BSDFType rayType = eReflected;

    float t;
    Vec3 n, ns;
    const Primitive* hit;

	float bsdfPdf = 1.0f;

    for (int i=0; i < maxDepth; ++i)
    {
        // find closest hit
        if (Trace(scene, Ray(rayOrigin, rayDir, rayTime), t, n, &hit))
        {	
			float outEta;
			Vec3 outAbsorption;

        	// index of refraction for transmission, 1.0 corresponds to air
			if (rayEta == 1.0f)
			{
        		outEta = hit->material.GetIndexOfRefraction();
				outAbsorption = Vec3(hit->material.absorption);
			}
			else
			{
				// returning to free space
				outEta = 1.0f;
				outAbsorption = 0.0f;
			}

			// update throughput based on absorption through the medium
			pathThroughput *= Color(Exp(-rayAbsorption*t), 1.0f);

#if 1
			
			if (i == 0)
			{
				// first trace is our only chance to add contribution from directly visible light sources        
				totalRadiance += hit->material.emission;
			}			
			else if (kBsdfSamples > 0)
			{
				// area pdf that this dir was already included by the light sampling from previous step
				float lightArea = LightArea(*hit);

				if (lightArea > 0.0f)
				{
					// convert to pdf with respect to solid angle
					float lightPdf = ((1.0f/lightArea)*t*t)/Abs(Dot(rayDir, n));

					// calculate weight for bsdf sampling
					int N = hit->lightSamples+kBsdfSamples;
					float cbsdf = kBsdfSamples/N;
					float clight = float(hit->lightSamples)/N;
					float weight = cbsdf*bsdfPdf/(cbsdf*bsdfPdf+ clight*lightPdf);
							
					Validate(weight);

					// specular paths have zero chance of being included by direct light sampling (zero pdf)
					if (rayType == eSpecular)
						weight = 1.0f;

					// pathThroughput already includes the bsdf pdf
					totalRadiance += weight*pathThroughput*hit->material.emission;
				}
			}

            // calculate a basis for this hit point
            Vec3 u, v;
            BasisFromVector(n, &u, &v);

            const Vec3 p = rayOrigin + rayDir*t;

/*
			if (hit->material.bump > 0.0f)
			{
				ns = FaceForward(EvaluateBumpNormal(n, p, hit->material.bumpMap, hit->material.bumpTile, hit->material.bump, rand), n);	
			}
			else
			{
				ns = n;
			}
*/

			// integrate direct light over hemisphere
			totalRadiance += pathThroughput*SampleLights(scene, *hit, rayEta, outEta, p, n, n, -rayDir, rayTime, rand);
#else
			
			// calculate a basis for this hit point
            Vec3 u, v;
            BasisFromVector(n, &u, &v);

            const Vec3 p = rayOrigin + rayDir*t + n*kRayEpsilon;

			totalRadiance += pathThroughput*hit->material.emission;

#endif

			// integrate indirect light by sampling BSDF
			Mat33 localFrame(u, v, n);

			Vec3 bsdfDir;
			BSDFType bsdfType;

			BSDFSample(hit->material, rayEta, outEta, p, Mat33(u,v,n), -rayDir, bsdfDir, bsdfPdf, bsdfType, rand);
			
            if (bsdfPdf <= 0.0f)
            	break;

			Validate(bsdfPdf);

            // reflectance
            Color f = BSDFEval(hit->material, rayEta, outEta, p, n, -rayDir, bsdfDir);

            // update ray medium if we are transmitting through the material
            if (Dot(bsdfDir, n) <= 0.0f)
			{
            	rayEta = outEta;
				rayAbsorption = outAbsorption;
			}

            // update throughput with primitive reflectance
            pathThroughput *= f * Abs(Dot(n, bsdfDir))/bsdfPdf;

            // update ray direction and type
            rayType = bsdfType;
			rayDir = bsdfDir;            
			rayOrigin = p + FaceForward(n, bsdfDir)*kRayEpsilon;
			
        }
        else
        {
            // hit nothing, sample sky dome and terminate         
            float weight = 1.0f;

        	if (scene.sky.probe.valid && i > 0 && rayType == eReflected)
        	{ 
        		// probability that this dir was already sampled by probe sampling
        		float skyPdf = ProbePdf(scene.sky.probe, rayDir);
				 
				int N = kProbeSamples+kBsdfSamples;
				float cbsdf = kBsdfSamples/N;
				float csky = float(kProbeSamples)/N;
			
				weight = cbsdf*bsdfPdf/(cbsdf*bsdfPdf+ csky*skyPdf);

				Validate(bsdfPdf);
				Validate(skyPdf);

			}

			Validate(weight);
		
       		totalRadiance += weight*scene.sky.Eval(rayDir)*pathThroughput; 
			break;
        }
    }

    return totalRadiance;
}

__device__ void AddSample(Color* output, int width, int height, float rasterX, float rasterY, float clamp, Filter filter, const Color& sample)
{
	switch (filter.type)
	{
		case eFilterBox:
		{
			int x = int(rasterX);
			int y = int(rasterY);

			output[y*width+x] += Color(sample.x, sample.y, sample.z, 1.0f);
			break;
		}
		case eFilterGaussian:
		{
			int startX = Max(0, int(rasterX - filter.width));
			int startY = Max(0, int(rasterY - filter.width));
			int endX = Min(int(rasterX + filter.width), width-1);
			int endY = Min(int(rasterY + filter.width), height-1);

			Color c =  ClampLength(sample, clamp);
			c.w = 1.0f;

			for (int x=startX; x <= endX; ++x)
			{
				for (int y=startY; y <= endY; ++y)
				{
					float w = filter.Eval(x-rasterX, y-rasterY);

					//output[(height-1-y)*width+x] += Color(Min(sample.x, clamp), Min(sample.y, clamp), Min(sample.z, clamp), 1.0f)*w;

					const int index = y*width+x;

					atomicAdd(&output[index].x, c.x*w);
					atomicAdd(&output[index].y, c.y*w);
					atomicAdd(&output[index].z, c.z*w);
					atomicAdd(&output[index].w, w);
				}
			}
		
			break;
		}
	};
}

__global__ void RenderGpu(GPUScene scene, Camera camera, CameraSampler sampler, Options options, int seed, Color* output)
{
	const int tx = blockIdx.x*blockDim.x;
	const int ty = blockIdx.y*blockDim.y;

	const int i = tx + threadIdx.x;
	const int j = ty + threadIdx.y;

	if (i < options.width && j < options.height)
	{
		Vec3 origin;
		Vec3 dir;

		// initialize a per-thread PRNG
		Random rand(i + j*options.width + seed);

		if (options.mode == eNormals)
		{
			sampler.GenerateRay(i, j, origin, dir);

			const Primitive* p;
			float t;
			Vec3 n;

			if (Trace(scene, Ray(origin, dir, 1.0f), t, n, &p))
			{
				n = n*0.5f+0.5f;
				output[j*options.width+i] = Color(n.x, n.y, n.z, 1.0f);
			}
			else
			{
				output[j*options.width+i] = Color(0.5f);
			}
		}
		else if (options.mode == ePathTrace)
		{
			const float time = rand.Randf(camera.shutterStart, camera.shutterEnd);
			const float fx = i + rand.Randf(-0.5f, 0.5f) + 0.5f;
			const float fy = j + rand.Randf(-0.5f, 0.5f) + 0.5f;

			Vec3 origin, dir;
			sampler.GenerateRay(fx, fy, origin, dir);

			//output[(height-1-j)*width+i] += PathTrace(*scene, origin, dir);
			Color sample = PathTrace(scene, origin, dir, time, options.maxDepth, rand);

			AddSample(output, options.width, options.height, fx, fy, options.clamp, options.filter, sample);
		}
	}
}

struct GpuRenderer : public Renderer
{
	Color* output = NULL;
	
	GPUScene sceneGPU;
	
	Random seed;

	// map id to geometry struct
	std::map<int, MeshGeometry> gpuMeshes;

	GpuRenderer(const Scene* s)
	{
		// build GPU primitive and light lists
		std::vector<Primitive> primitives;		
		std::vector<Primitive> lights;

		for (int i=0; i < s->primitives.size(); ++i)
		{
			Primitive primitive = s->primitives[i];

			// if mesh primitive then copy to the GPU
			if (primitive.type == eMesh)
			{
				// see if we have already uploaded the mesh to the GPU
				if (gpuMeshes.find(primitive.mesh.id) == gpuMeshes.end())
				{
					MeshGeometry geo = CreateGPUMesh(primitive.mesh);
					gpuMeshes[geo.id] = geo;

					// replace CPU mesh with GPU copy
					primitive.mesh = geo;
				}
			}

			if (primitive.material.bump > 0.0f)
			{
				primitive.material.bumpMap = CreateGPUTexture(primitive.material.bumpMap);
			}
			
			// create explicit list of light primitives
			if (primitive.lightSamples)
			{
				lights.push_back(primitive);
			}

			primitives.push_back(primitive);
		}

		// upload to the GPU
		sceneGPU.numPrimitives = primitives.size();
		sceneGPU.numLights = lights.size();

		if (sceneGPU.numLights > 0)
		{
			hipMalloc(&sceneGPU.lights, sizeof(Primitive)*lights.size());
			hipMemcpy(sceneGPU.lights, &lights[0], sizeof(Primitive)*lights.size(), hipMemcpyHostToDevice);
		}

		if (sceneGPU.numPrimitives > 0)
		{
			hipMalloc(&sceneGPU.primitives, sizeof(Primitive)*primitives.size());
			hipMemcpy(sceneGPU.primitives, &primitives[0], sizeof(Primitive)*primitives.size(), hipMemcpyHostToDevice);
		}

		// copy sky and probe texture
		sceneGPU.sky = CreateGPUSky(s->sky);
	}

	virtual ~GpuRenderer()
	{
		hipFree(output);
		hipFree(sceneGPU.primitives);
		hipFree(sceneGPU.lights);
	}
	
	void Init(int width, int height)
	{
		hipFree(output);
		hipMalloc(&output, sizeof(Color)*width*height);
		hipMemset(output, 0, sizeof(Color)*width*height);
	}

	void Render(const Camera& camera, const Options& options, Color* outputHost)
	{
		// create a sampler for the camera
		CameraSampler sampler(
			Transform(camera.position, camera.rotation),
			camera.fov, 
			0.001f,
			1.0f,
			options.width,
			options.height);


		// assign threads in non-square tiles to match warp width
		const int blockWidth = 32;
		const int blockHeight = 8;

		const int gridWidth = (options.width + blockWidth - 1)/blockWidth;
		const int gridHeight = (options.height + blockHeight - 1)/blockHeight;

		dim3 blockDim(blockWidth, blockHeight);
		dim3 gridDim(gridWidth, gridHeight);

		RenderGpu<<<gridDim, blockDim>>>(sceneGPU, camera, sampler, options, seed.Rand(), output);

		// copy back to output
		hipMemcpy(outputHost, output, sizeof(Color)*options.width*options.height, hipMemcpyDeviceToHost);
	}
};


Renderer* CreateGpuRenderer(const Scene* s)
{
	return new GpuRenderer(s);
}
