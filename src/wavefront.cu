#include "hip/hip_runtime.h"
#include "maths.h"
#include "render.h"
#include "util.h"
#include "disney.h"
#include "bvh.h"

#include <map>

struct GPUScene
{
	Primitive* primitives;
	int numPrimitives;

	Primitive* lights;
	int numLights;

	Sky sky;

	BVH bvh;
};

#define kBsdfSamples 1.0f
#define kProbeSamples 1.0f
#define kRayEpsilon 0.0001f

#define LAUNCH_BOUNDS __launch_bounds__(256, 4)

__device__ inline int getGlobalIndex()
{
	int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
	int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
	return threadId;
}

// create a texture object from memory and store it in a 64-bit pointer
void CreateIntTexture(int** deviceBuffer, const int* hostBuffer, int sizeInBytes)
{
	int* buffer;
	hipMalloc(&buffer, sizeInBytes);
	hipMemcpy(buffer, hostBuffer, sizeInBytes, hipMemcpyHostToDevice);

#if USE_TEXTURES

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = (void*)buffer;
	resDesc.res.linear.desc.f = hipChannelFormatKindSigned;
	resDesc.res.linear.desc.x = 32; // bits per channel
	resDesc.res.linear.sizeInBytes = sizeInBytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	// cast to pointer
	*deviceBuffer = (int*)tex;
#else

	*deviceBuffer = buffer;

#endif
}

// create a texture object from memory and store it in a 64-bit pointer
void CreateFloatTexture(float** deviceBuffer, const float* hostBuffer, int sizeInBytes)
{
	float* buffer;
	hipMalloc(&buffer, sizeInBytes);
	hipMemcpy(buffer, hostBuffer, sizeInBytes, hipMemcpyHostToDevice);

#if USE_TEXTURES

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = (void*)buffer;
	resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc.res.linear.desc.x = 32; // bits per channel
	resDesc.res.linear.sizeInBytes = sizeInBytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	// cast to pointer
	*deviceBuffer = (float*)tex;

#else

	*deviceBuffer = buffer;

#endif
}

// create a texture object from memory and store it in a 64-bit pointer
void CreateVec4Texture(Vec4** deviceBuffer, const Vec4* hostBuffer, int sizeInBytes)
{
	Vec4* buffer;
	hipMalloc(&buffer, sizeInBytes);
	hipMemcpy(buffer, hostBuffer, sizeInBytes, hipMemcpyHostToDevice);

#if USE_TEXTURES

	// create texture object
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = (void*)buffer;
	resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
	resDesc.res.linear.desc.x = 32; // bits per channel
	resDesc.res.linear.desc.y = 32; // bits per channel
	resDesc.res.linear.desc.z = 32; // bits per channel
	resDesc.res.linear.desc.w = 32; // bits per channel
	resDesc.res.linear.sizeInBytes = sizeInBytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;

	hipTextureObject_t tex;
	hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

	// cast to pointer
	*deviceBuffer = (Vec4*)tex;

#else

	*deviceBuffer = buffer;

#endif

}


MeshGeometry CreateGPUMesh(const MeshGeometry& hostMesh)
{
	const int numVertices = hostMesh.numVertices;
	const int numIndices = hostMesh.numIndices;
	const int numNodes = hostMesh.numNodes;

	
	MeshGeometry gpuMesh;

#if USE_TEXTURES
	
	// expand positions out to vec4
	std::vector<Vec4> positions;
	std::vector<Vec4> normals;

	for (int i=0; i < numVertices; ++i)
	{
		positions.push_back(Vec4(hostMesh.positions[i], 1.0f));
		normals.push_back(Vec4(hostMesh.normals[i], 0.0f));
	}

	CreateVec4Texture((Vec4**)&gpuMesh.positions, (Vec4*)&positions[0], sizeof(Vec4)*numVertices);
	CreateVec4Texture((Vec4**)&gpuMesh.normals, (Vec4*)&normals[0], sizeof(Vec4)*numVertices);

#else
	CreateFloatTexture((float**)&gpuMesh.positions, (float*)&hostMesh.positions[0], sizeof(Vec3)*numVertices);
	CreateFloatTexture((float**)&gpuMesh.normals, (float*)&hostMesh.normals[0], sizeof(Vec3)*numVertices);

#endif

	CreateIntTexture((int**)&gpuMesh.indices, (int*)&hostMesh.indices[0], sizeof(int)*numIndices);
	

	/*
	hipMalloc((Vec3**)&gpuMesh.positions, sizeof(Vec3)*numVertices);
	hipMemcpy((Vec3*)gpuMesh.positions, &hostMesh.positions[0], sizeof(Vec3)*numVertices, hipMemcpyHostToDevice);

	hipMalloc((Vec3**)&gpuMesh.normals, sizeof(Vec3)*numVertices);
	hipMemcpy((Vec3*)gpuMesh.normals, &hostMesh.normals[0], sizeof(Vec3)*numVertices, hipMemcpyHostToDevice);

	hipMalloc((int**)&gpuMesh.indices, sizeof(int)*numIndices);
	hipMemcpy((int*)gpuMesh.indices, &hostMesh.indices[0], sizeof(int)*numIndices, hipMemcpyHostToDevice);
	*/
	//hipMalloc((BVHNode**)&gpuMesh.nodes, sizeof(BVHNode)*numNodes);
	//hipMemcpy((BVHNode*)gpuMesh.nodes, &hostMesh.nodes[0], sizeof(BVHNode)*numNodes, hipMemcpyHostToDevice);

	CreateVec4Texture((Vec4**)&gpuMesh.nodes, (Vec4*)&hostMesh.nodes[0], sizeof(BVHNode)*numNodes);
	
	hipMalloc((float**)&gpuMesh.cdf, sizeof(float)*numIndices/3);
	hipMemcpy((float*)gpuMesh.cdf, &hostMesh.cdf[0], sizeof(float)*numIndices/3, hipMemcpyHostToDevice);
	
	gpuMesh.numIndices = numIndices;
	gpuMesh.numVertices = numVertices;
	gpuMesh.numNodes = numNodes;
	gpuMesh.area = hostMesh.area;

	return gpuMesh;

}

void DestroyGPUMesh(const MeshGeometry& m)
{

}

Texture CreateGPUTexture(const Texture& tex)
{
	const int numTexels = tex.width*tex.height*tex.depth;
	
	Texture gpuTex = tex;

	hipMalloc((void**)&gpuTex.data, sizeof(float)*numTexels);
	hipMemcpy(gpuTex.data, tex.data, sizeof(float)*numTexels, hipMemcpyHostToDevice);

	return gpuTex;
}

Sky CreateGPUSky(const Sky& sky)
{
	Sky gpuSky = sky;

	// copy probe
	if (sky.probe.valid)
	{
		const int numPixels = sky.probe.width*sky.probe.height;

		// copy pixel data
		CreateVec4Texture((Vec4**)&gpuSky.probe.data, sky.probe.data, numPixels*sizeof(float)*4);

		// copy cdf tables
		CreateFloatTexture((float**)&gpuSky.probe.cdfValuesX, sky.probe.cdfValuesX, numPixels*sizeof(float));
		CreateFloatTexture((float**)&gpuSky.probe.pdfValuesX, sky.probe.pdfValuesX, numPixels*sizeof(float));

		CreateFloatTexture((float**)&gpuSky.probe.cdfValuesY, sky.probe.cdfValuesY, sky.probe.height*sizeof(float));
		CreateFloatTexture((float**)&gpuSky.probe.pdfValuesY, sky.probe.pdfValuesY, sky.probe.height*sizeof(float));
	}

	return gpuSky;
}

void DestroyGPUSky(const Sky& gpuSky)
{
	if (gpuSky.probe.valid)
	{
		// todo
	}
}


#if 1


inline __device__ bool Trace(const GPUScene& scene, const Vec3& rayOrigin, const Vec3& rayDir, float rayTime, float& outT, Vec3& outNormal, const Primitive** RESTRICT outPrimitive)
{
	int stack[64];
	stack[0] = 0;

	unsigned int count = 1;

	Vec3 dir, rcpDir;
	Vec3 origin;
	
	rcpDir.x = 1.0f/rayDir.x;
	rcpDir.y = 1.0f/rayDir.y;
	rcpDir.z = 1.0f/rayDir.z;
	origin = rayOrigin;
	dir = rayDir;

	const BVHNode* RESTRICT root = scene.bvh.nodes;

	MeshGeometry mesh;
	int primitiveIndex = -1;

	float closestT = FLT_MAX;
	//float closestU;
	float closestV;
	float closestW;

	Vec3 closestNormal;
	int closestPrimitive = -1;
	int closestTri;

	while(count)
	{
		const int nodeIndex = stack[--count];

		if (nodeIndex < 0)
		{
			// reset to scene bvh dir and address
			rcpDir.x = 1.0f/rayDir.x;
			rcpDir.y = 1.0f/rayDir.y;
			rcpDir.z = 1.0f/rayDir.z;
			origin = rayOrigin;
			dir = rayDir;
			root = scene.bvh.nodes;
			primitiveIndex = -1;

			continue;
		}

		BVHNode node = fetchNode(root, nodeIndex);

		int leftIndex = node.leftIndex;
		int rightIndex = node.rightIndex;

		if (node.leaf)
		{
			if (primitiveIndex < 0)
			{
				const Primitive& p = scene.primitives[leftIndex];

				Transform transform = InterpolateTransform(p.startTransform, p.endTransform, rayTime);

				switch (p.type)
				{
					case eSphere:
					{
						float minT, maxT;
						Vec3 n;

						bool hit = IntersectRaySphere(transform.p, p.sphere.radius*transform.s, origin, dir, minT, maxT, &n);

						if (hit && minT < closestT)
						{
							closestT = minT;
							closestNormal = n;
							closestPrimitive = leftIndex;
						}
						break;
					}
					case ePlane:
					{
						float t;
						bool hit = IntersectRayPlane(origin, dir, (const Vec4&)p.plane, t);
			
						if (hit && t < closestT)
						{
							closestT = t;
							closestNormal = (const Vec3&)p.plane;							
							closestPrimitive = leftIndex;
						}

						break;
					}
					case eMesh:
					{
						// push a back-tracking marker in the stack
						stack[count++] = -1;

						// push root of the mesh bvh
						stack[count++] = 0;

						// transform ray to primitive local space
						origin = InverseTransformPoint(transform, rayOrigin);					
						dir = InverseTransformVector(transform, rayDir);

						rcpDir.x = 1.0f/dir.x;
						rcpDir.y = 1.0f/dir.y;
						rcpDir.z = 1.0f/dir.z;				
				
						// set bvh and mesh sources
						root = p.mesh.nodes;
						mesh = p.mesh;

						primitiveIndex = leftIndex;
						break;
					}
				};
			}
			else
			{
				// mesh mode
				int i0 = fetchInt(mesh.indices, leftIndex*3+0);
				int i1 = fetchInt(mesh.indices, leftIndex*3+1);
				int i2 = fetchInt(mesh.indices, leftIndex*3+2);

				const Vec3 a = fetchVec3(mesh.positions, i0);
				const Vec3 b = fetchVec3(mesh.positions, i1);
				const Vec3 c = fetchVec3(mesh.positions, i2);

				float t, u, v, w;
				float sign;
				Vec3 n;
				//if (IntersectRayTri(rayOrigin, rayDir, a, b, c, t, u, v, w, &n))
				if (IntersectRayTriTwoSided(origin, dir, a, b, c, t, u, v, w, sign, &n))
				{
					if (t > 0.0f && t < closestT)
					{
						closestT = t;
						//closestU = u;
						closestV = v;
						closestW = w;

						closestTri = leftIndex;
						closestNormal = n*sign;						
						closestPrimitive = primitiveIndex;
					}
				}
			}
		}
		else
		{
			// check children
			BVHNode left = fetchNode(root, leftIndex);
			BVHNode right = fetchNode(root, rightIndex);

			float tLeft;
			bool hitLeft = IntersectRayAABBFast(origin, rcpDir, left.bounds.lower, left.bounds.upper, tLeft);// && tLeft < closestT;

			float tRight;
			bool hitRight = IntersectRayAABBFast(origin, rcpDir, right.bounds.lower, right.bounds.upper, tRight);// && tRight < closestT;

			// traverse closest first
			if (hitLeft && hitRight && (tLeft < tRight))
			{
				//Swap(leftIndex, rightIndex);
			}

			if (hitLeft)
				stack[count++] = leftIndex;

			if (hitRight)
				stack[count++] = rightIndex;			
		}
	}

	
	if (closestPrimitive >= 0)
	{
		const Primitive& p = scene.primitives[closestPrimitive];

		if (p.type == eMesh)
		{
			Transform transform = InterpolateTransform(p.startTransform, p.endTransform, rayTime);

			// interpolate vertex normals
			int i0 = fetchInt(p.mesh.indices, closestTri*3+0);
			int i1 = fetchInt(p.mesh.indices, closestTri*3+1);
			int i2 = fetchInt(p.mesh.indices, closestTri*3+2);

			const Vec3 n1 = fetchVec3(p.mesh.normals, i0);
			const Vec3 n2 = fetchVec3(p.mesh.normals, i1);
			const Vec3 n3 = fetchVec3(p.mesh.normals, i2);

			Vec3 smoothNormal = (1.0f-closestV-closestW)*n1 + closestV*n2 + closestW*n3;

			// ensure smooth normal lies on the same side of the geometric normal
			if (Dot(smoothNormal, closestNormal) < 0.0f)
				smoothNormal *= -1.0f;

			closestNormal = SafeNormalize(TransformVector(transform, smoothNormal), closestNormal);
		}

		outT = closestT;
		outNormal = FaceForward(closestNormal, -rayDir);
		
		if (outPrimitive)
			*outPrimitive = &p;

		return true;
	}
	else
	{
		// no hit
		return false;
	}
}

#else

// trace a ray against the scene returning the closest intersection
inline __device__ bool Trace(const GPUScene& scene, const Vec3& rayOrigin, const Vec3& rayDir, float rayTime, float& outT, Vec3& outNormal, const Primitive** outPrimitive)
{

#if 0

	struct Callback
	{
		float minT;
		Vec3 closestNormal;
		const Primitive* closestPrimitive;

		const Ray& ray;
		const GPUScene& scene;

		CUDA_CALLABLE inline Callback(const GPUScene& s, const Ray& r) : minT(REAL_MAX), closestPrimitive(NULL), ray(r), scene(s)
		{

		}
		
		CUDA_CALLABLE inline void operator()(int index)
		{
			float t;
			Vec3 n, ns;

			const Primitive& primitive = scene.primitives[index];

			if (PrimitiveIntersect(primitive, ray, t, &n))
			{
				if (t < minT && t > 0.0f)
				{
					minT = t;
					closestPrimitive = &primitive;
					closestNormal = n;
				}
			}			
		}
	};

	Callback callback(scene, ray);
	QueryBVH(callback, scene.bvh.nodes, ray.origin, ray.dir);

	outT = callback.minT;		
	outNormal = FaceForward(callback.closestNormal, -ray.dir);
	
	if (outPrimitive)
		*outPrimitive = callback.closestPrimitive;

	return callback.closestPrimitive != NULL;
	
#else

	float minT = REAL_MAX;
	const Primitive* closestPrimitive = NULL;
	Vec3 closestNormal(0.0f);

	for (int i=0; i < scene.numPrimitives; ++i)
	{
		const Primitive& primitive = scene.primitives[i];

		float t;
		Vec3 n;

		if (PrimitiveIntersect(primitive, Ray(rayOrigin, rayDir, rayTime), t, &n))
		{
			if (t < minT && t > 0.0f)
			{
				minT = t;
				closestPrimitive = &primitive;
				closestNormal = n;
			}
		}
	}
	
	outT = minT;		
	outNormal = FaceForward(closestNormal, -rayDir);
	
	if (outPrimitive)
		*outPrimitive = closestPrimitive;

	return closestPrimitive != NULL;

#endif

}

#endif


__device__ inline float SampleTexture(const Texture& map, int i, int j, int k)
{
	int x = int(Abs(i))%map.width;
	int y = int(Abs(j))%map.height;
	int z = int(Abs(k))%map.depth;
	
	return map.data[z*map.width*map.height + y*map.width + x];
}


__device__ inline float LinearInterp(const Texture& map, const Vec3& pos) 
{
	int i = floorf(pos.x*map.width);
	int j = floorf(pos.y*map.height);
	int k = floorf(pos.z*map.depth);
		
	// trilinear interpolation
	float tx = pos.x*map.width-i;
	float ty = pos.y*map.height-j;
	float tz = pos.z*map.depth-k;
		
	float a = Lerp(SampleTexture(map, i, j, k), SampleTexture(map, i, j, k+1), tz);
	float b = Lerp(SampleTexture(map, i+1, j, k), SampleTexture(map, i+1, j, k+1), tz);
	float c = Lerp(SampleTexture(map, i, j+1, k), SampleTexture(map, i, j+1, k+1), tz);		
	float d = Lerp(SampleTexture(map, i+1, j+1, k), SampleTexture(map, i+1, j+1, k+1), tz);
		
	float e = Lerp(a, b, tx);
	float f = Lerp(c, d, tx);
		
	float g = Lerp(e, f, ty);
		
	return g;
}

__device__ inline Vec3 EvaluateBumpNormal(const Vec3& surfaceNormal, const Vec3& surfacePos, const Texture& bumpMap, const Vec3& bumpTile, float bumpStrength, Random& rand)
{
	Vec3 u, v;
	BasisFromVector(surfaceNormal, &u, &v);

	float eps = 0.01f;

	Vec3 dpdu = u + bumpStrength*surfaceNormal*(LinearInterp(bumpMap, bumpTile*(surfacePos)+u*eps) - LinearInterp(bumpMap, bumpTile*surfacePos))/eps;
	Vec3 dpdv = v + bumpStrength*surfaceNormal*(LinearInterp(bumpMap, bumpTile*(surfacePos)+v*eps) - LinearInterp(bumpMap, bumpTile*surfacePos))/eps;

	return SafeNormalize(Cross(dpdu, dpdv), surfaceNormal);
}



__device__ inline Vec3 SampleLights(const GPUScene& scene, const Primitive& surfacePrimitive, float etaI, float etaO, const Vec3& surfacePos, const Vec3& surfaceNormal, const Vec3& shadingNormal, const Vec3& wo, float time, Random& rand)
{	
	Vec3 sum(0.0f);
	
	if (scene.sky.probe.valid)
	{
		for (int i=0; i < kProbeSamples; ++i)
		{

			Vec3 skyColor;
			float skyPdf;
			Vec3 wi;

			ProbeSample(scene.sky.probe, wi, skyColor, skyPdf, rand);
			
			
			//wi = UniformSampleSphere(rand);
			//skyColor = ProbeEval(scene.sky.probe, ProbeDirToUV(wi));
			//skyPdf = 0.5f*kInv2Pi;
						
			
			//if (Dot(wi, surfaceNormal) <= 0.0f)
//				continue;

			// check if occluded
			float t;
			Vec3 n;
			
			if (Trace(scene, surfacePos + FaceForward(surfaceNormal, wi)*kRayEpsilon, wi, time, t, n, NULL) == false)
			{
				float bsdfPdf = BSDFPdf(surfacePrimitive.material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
				Vec3 f = BSDFEval(surfacePrimitive.material, etaI, etaO, surfacePos, surfaceNormal, wo, wi);
				
				if (bsdfPdf > 0.0f)
				{
					int N = kProbeSamples+kBsdfSamples;
					float cbsdf = kBsdfSamples/N;
					float csky = float(kProbeSamples)/N;
					float weight = csky*skyPdf/(cbsdf*bsdfPdf + csky*skyPdf);

					Validate(weight);

					if (weight > 0.0f)
						sum += weight*skyColor*f*Abs(Dot(wi, surfaceNormal))/skyPdf;
				}
			}
		}

		if (kProbeSamples > 0)
			sum /= float(kProbeSamples);
	}

	for (int i=0; i < scene.numLights; ++i)
	{
		// assume all lights are area lights for now
		const Primitive& lightPrimitive = scene.lights[i];

		Vec3 L(0.0f);

		int numSamples = lightPrimitive.lightSamples;

		if (numSamples == 0)
			continue;

		for (int s=0; s < numSamples; ++s)
		{
			// sample light source
			Vec3 lightPos;
			Vec3 lightNormal;

			PrimitiveSample(lightPrimitive, time, lightPos, lightNormal, rand);
			
			Vec3 wi = lightPos-surfacePos;
			
			float dSq = LengthSq(wi);
			wi /= sqrtf(dSq);


			// light is behind surface
			//if (Dot(wi, surfaceNormal) <= 0.0f)
				//continue; 				

			// surface is behind light
			if (Dot(wi, lightNormal) >= 0.0f)
				continue;

			// check visibility
			float t;
			Vec3 n;			
			if (Trace(scene, surfacePos + FaceForward(surfaceNormal, wi)*kRayEpsilon, wi, time, t, n, NULL))			
			{
				float tSq = t*t;

				// if our next hit was further than distance to light then accept
				// sample, this works for portal sampling where you have a large light
				// that you sample through a small window
				const float kTolerance = 1.e-2f;

				if (fabsf(t - sqrtf(dSq)) <= kTolerance)
				{				
					const float nl = Abs(Dot(lightNormal, wi));

					// light pdf with respect to area and convert to pdf with respect to solid angle
					float lightArea = PrimitiveArea(lightPrimitive);
					float lightPdf = ((1.0f/lightArea)*tSq)/nl;

					// bsdf pdf for light's direction
					float bsdfPdf = BSDFPdf(surfacePrimitive.material, etaI, etaO, surfacePos, shadingNormal, wo, wi);
					Vec3 f = BSDFEval(surfacePrimitive.material, etaI, etaO, surfacePos, shadingNormal, wo, wi);

					// this branch is only necessary to exclude specular paths from light sampling (always have zero brdf)
					// todo: make BSDFEval alwasy return zero for pure specular paths and roll specular eval into BSDFSample()
					if (bsdfPdf > 0.0f)
					{
						// calculate relative weighting of the light and bsdf sampling
						int N = lightPrimitive.lightSamples+kBsdfSamples;
						float cbsdf = kBsdfSamples/N;
						float clight = float(lightPrimitive.lightSamples)/N;
						float weight = clight*lightPdf/(cbsdf*bsdfPdf + clight*lightPdf);
						
						L += weight*f*lightPrimitive.material.emission*(Abs(Dot(wi, shadingNormal))/Max(1.e-3f, lightPdf));
					}
				}
			}
		}
	
		sum += L * (1.0f/numSamples);
	}

	return sum;
}


struct Tile
{
	int x;
	int y;
	int width;
	int height;
};

enum PathMode
{
	ePathGenerate,
	ePathAdvance,
	ePathProbeSample,
	ePathLightSample,
	ePathBsdfSample,
	ePathTerminate,
	ePathDisabled,
};


struct PathState
{		
	Vec3* __restrict__ rayOrigin;
	Vec3* __restrict__ rayDir;
	float* __restrict__ rayTime;

	Vec3* __restrict__ pos;
	Vec3* __restrict__ normal;

	int* __restrict__ depth;

	Vec3* __restrict__ pathThroughput;
	Vec3* __restrict__ absorption;
	const Primitive** __restrict__ primitive;

	Vec3* __restrict__ totalRadiance;

	float* __restrict__ etaI;
	float* __restrict__ etaO;

	PathMode* __restrict__ mode;

	// pdf from last brdf sampling
	float* __restrict__ bsdfPdf;
	BSDFType* __restrict__ bsdfType;

	// sample coordinate
	float* __restrict__ rasterX;
	float* __restrict__ rasterY;

	Random* __restrict__ rand;
};

template <typename T>
void Alloc(T** ptr, int num)
{
	hipMalloc(ptr, sizeof(T)*num);
	hipMemset(*ptr, 0, sizeof(T)*num);
}

PathState AllocatePaths(int num)
{
	PathState state;

	Alloc(&state.rayOrigin, num);
	Alloc(&state.rayDir, num);
	Alloc(&state.rayTime, num);

	Alloc(&state.pos, num);
	Alloc(&state.normal, num);

	Alloc(&state.depth, num);

	Alloc(&state.pathThroughput, num);
	Alloc(&state.absorption, num);
	Alloc(&state.primitive, num);
	Alloc(&state.totalRadiance, num);

	Alloc(&state.etaI, num);
	Alloc(&state.etaO, num);

	Alloc(&state.mode, num);

	Alloc(&state.bsdfPdf, num);
	Alloc(&state.bsdfType, num);

	Alloc(&state.rasterX, num);
	Alloc(&state.rasterY, num);

	Alloc(&state.rand, num);

	return state;
}

void FreePaths(PathState state)
{
	// todo:
}




LAUNCH_BOUNDS
__global__ void TerminatePaths(Color* output, Options options, PathState paths, int numPaths)
{
	const int i = getGlobalIndex();

	{
		if (paths.mode[i] != ePathDisabled)
		{
			float rasterX = paths.rasterX[i];
			float rasterY = paths.rasterY[i];

			Vec3 sample = paths.totalRadiance[i];

			// sample = paths[i].normal*0.5f + 0.5f;

			int width = options.width;
			int height = options.height;

			Filter filter = options.filter;

			switch (filter.type)
			{
				case eFilterBox:
				{
					int x = Clamp(int(rasterX), 0, width-1);
					int y = Clamp(int(rasterY), 0, height-1);

					output[y*width+x] += Color(sample.x, sample.y, sample.z, 1.0f);
					break;
				}
				case eFilterGaussian:
				{
					int startX = Max(0, int(rasterX - filter.width));
					int startY = Max(0, int(rasterY - filter.width));
					int endX = Min(int(rasterX + filter.width), width-1);
					int endY = Min(int(rasterY + filter.width), height-1);

					Vec3 c =  ClampLength(sample, options.clamp);

					for (int x=startX; x <= endX; ++x)
					{
						for (int y=startY; y <= endY; ++y)
						{
							float w = filter.Eval(x-rasterX, y-rasterY);

							//output[(height-1-y)*width+x] += Vec3(Min(sample.x, clamp), Min(sample.y, clamp), Min(sample.z, clamp), 1.0f)*w;

							const int index = y*width+x;

							atomicAdd(&output[index].x, c.x*w);
							atomicAdd(&output[index].y, c.y*w);
							atomicAdd(&output[index].z, c.z*w);
							atomicAdd(&output[index].w, w);
						}
					}
		
					break;
				}
			};
		}

		paths.mode[i] = ePathGenerate;
	}
}

LAUNCH_BOUNDS
__global__ void SampleLights(GPUScene scene, PathState paths, int numPaths)
{
	const int i = getGlobalIndex();

	{
		if (paths.mode[i] == ePathLightSample)
		{
        	// calculate a basis for this hit point
        	const Primitive* hit = paths.primitive[i];        	
        	
        	float etaI = paths.etaI[i];
        	float etaO = paths.etaO[i];

			const Vec3 rayDir = paths.rayDir[i];
            float rayTime = paths.rayTime[i];

            const Vec3 p = paths.pos[i];
            const Vec3 n = paths.normal[i];

			// integrate direct light over hemisphere
			paths.totalRadiance[i] += paths.pathThroughput[i]*SampleLights(scene, *hit, etaI, etaO, p, n, n, -rayDir, rayTime, paths.rand[i]);			

			paths.mode[i] = ePathBsdfSample;		
		}
	}
}

LAUNCH_BOUNDS
__global__ void SampleBsdfs(PathState paths, int numPaths)
{
	const int i = getGlobalIndex();

	{
		if (paths.mode[i] == ePathBsdfSample)
		{	
			const Vec3 p = paths.pos[i];
			const Vec3 n = paths.normal[i];

			const Vec3 rayDir = paths.rayDir[i];

			const Primitive* hit = paths.primitive[i];

			Random& rand = paths.rand[i];

			float etaI = paths.etaI[i];
			float etaO = paths.etaO[i];

			// integrate indirect light by sampling BRDF
            Vec3 u, v;
            BasisFromVector(n, &u, &v);

			Vec3 bsdfDir;
			BSDFType bsdfType;
			float bsdfPdf;

			BSDFSample(hit->material, etaI, etaO, p, u, v, n, -rayDir, bsdfDir, bsdfPdf, bsdfType, rand);

            if (bsdfPdf <= 0.0f)
           	{
           		paths.mode[i] = ePathTerminate;
           	}
           	else
           	{
	            // reflectance
	            Vec3 f = BSDFEval(hit->material, etaI, etaO, p, n, -rayDir, bsdfDir);

	            // update ray medium if we are transmitting through the material
	            if (Dot(bsdfDir, n) <= 0.0f)
	            {
	            	paths.etaI[i] = etaO;
	            	paths.bsdfType[i] = eTransmitted;
					
	            	if (etaI != 1.0f)
	            	{
	            		// entering a medium, update the aborption (assume zero in air)
						paths.absorption[i] = hit->material.absorption;
					}
	            }
	            else
	            {
	            	paths.bsdfType[i] = eReflected;
	            }

	            // update throughput with primitive reflectance
	            paths.pathThroughput[i] *= f * Abs(Dot(n, bsdfDir))/bsdfPdf;
	            paths.bsdfPdf[i] = bsdfPdf;
	            paths.bsdfType[i] = bsdfType;
	            paths.rayDir[i] = bsdfDir;
	            paths.rayOrigin[i] = p + FaceForward(n, bsdfDir)*kRayEpsilon;
	            paths.mode[i] = ePathAdvance;

	        }
        }
    }
}

LAUNCH_BOUNDS
__global__ void SampleProbes(PathState paths, int numPaths)
{

}

LAUNCH_BOUNDS
__global__ void AdvancePaths(GPUScene scene, PathState paths, int numPaths)
{
	const int i = getGlobalIndex();

	{
		if (paths.mode[i] == ePathAdvance)
		{
			Vec3 rayOrigin = paths.rayOrigin[i];
			Vec3 rayDir = paths.rayDir[i];
			float rayTime = paths.rayTime[i];
			float etaI = paths.etaI[i];

			Vec3 pathThroughput = paths.pathThroughput[i];

			Vec3 n;
			float t;
			const Primitive* hit;

	        // find closest hit
	        if (Trace(scene, rayOrigin, rayDir, rayTime, t, n, &hit))
	        {	
				float etaO;

	        	// index of refraction for transmission, 1.0 corresponds to air
				if (etaI == 1.0f)
				{
	        		etaO = hit->material.GetIndexOfRefraction();
				}
				else
				{
					// returning to free space
					etaO = 1.0f;
				}

				pathThroughput *= Exp(-paths.absorption[i]*t);

				if (paths.depth[i] == 0)
				{
					// first trace is our only chance to add contribution from directly visible light sources        
					paths.totalRadiance[i] += hit->material.emission;
				}			
				else if (kBsdfSamples > 0)
				{
					// area pdf that this dir was already included by the light sampling from previous step
					float lightArea = PrimitiveArea(*hit);

					if (lightArea > 0.0f)
					{
						// convert to pdf with respect to solid angle
						float lightPdf = ((1.0f/lightArea)*t*t)/Clamp(Dot(-rayDir, n), 1.e-3f, 1.0f);

						// calculate weight for bsdf sampling
						int N = hit->lightSamples+kBsdfSamples;
						float cbsdf = kBsdfSamples/N;
						float clight = float(hit->lightSamples)/N;
						float weight = cbsdf*paths.bsdfPdf[i]/(cbsdf*paths.bsdfPdf[i] + clight*lightPdf);
						
						// specular paths have zero chance of being included by direct light sampling (zero pdf)
						if (paths.bsdfType[i] == eSpecular)
							weight = 1.0f;

						// pathThroughput already includes the bsdf pdf
						paths.totalRadiance[i] += weight*pathThroughput*hit->material.emission;
					}
				}

				// terminate ray if we hit a light source
				if (hit->lightSamples)
				{
					paths.mode[i] = ePathTerminate;
				}
				else
				{
					// update throughput based on absorption through the medium
					paths.pos[i] = rayOrigin + rayDir*t;
					paths.normal[i] = n;
					paths.primitive[i] = hit;
					paths.etaO[i] = etaO;
					paths.pathThroughput[i] = pathThroughput;
					paths.depth[i] += 1;

					paths.mode[i] = ePathLightSample;
				}
			}
			else
			{
				// todo: sky 

				// no hit, terminate path
				paths.mode[i] = ePathTerminate;
			}
		}
	}
}

LAUNCH_BOUNDS
__global__ void GeneratePaths(Camera camera, CameraSampler sampler, Tile tile, int seed, PathState paths, int numPaths)
{
	const int tx = blockIdx.x*blockDim.x;
	const int ty = blockIdx.y*blockDim.y;

	const int x = tx + threadIdx.x + tile.x;
	const int y = ty + threadIdx.y + tile.y;

	const int i = getGlobalIndex();

	{
		if (paths.mode[i] == ePathGenerate || paths.mode[i] == ePathDisabled || paths.mode[i] == ePathTerminate)
		{
			// if we're inside the tile
			if (threadIdx.x < tile.width && threadIdx.y < tile.height)
			{
				Random rand(i + tile.y*tile.width + tile.x + seed);

				// offset
				//float x, y, t;
				//StratifiedSample2D(i, tile.width, tile.height, rand, x, y);

				float t;
				StratifiedSample1D(i, 64, rand, t);

				// shutter time
				float time = Lerp(camera.shutterStart, camera.shutterEnd, t);
				
				//float px = tile.x + x*tile.width;
				//float py = tile.y + y*tile.height;
				float px = x + rand.Randf(-0.5f, 0.5f);
				float py = y + rand.Randf(-0.5f, 0.5f);

				Vec3 origin, dir;
				sampler.GenerateRay(px, py, origin, dir);

				// advance paths
				paths.depth[i] = 0;
				paths.rayOrigin[i] = origin;
				paths.rayDir[i] = dir;
				paths.rayTime[i] = time;
				paths.mode[i] = ePathAdvance;
				paths.rand[i] = rand;
				paths.totalRadiance[i] = 0.0f;
				paths.pathThroughput[i] = 1.0f;
				paths.etaI[i] = 1.0f;
				paths.bsdfType[i] = eReflected;
				paths.bsdfPdf[i] = 1.0f;
				paths.rasterX[i] = px;
				paths.rasterY[i] = py;

			}
			else
			{
				paths.mode[i] = ePathDisabled;
			}
		}
	}
}

//LAUNCH_BOUNDS
__global__ void VisualizeNormals(GPUScene scene, PathState paths, int numPaths)
{
	const int i = getGlobalIndex();

	if (i < numPaths)
	{
		Vec3 rayOrigin = paths.rayOrigin[i];
		Vec3 rayDir = paths.rayDir[i];

		Vec3 n;
		float t;
		
	    // find closest hit
	    if (Trace(scene, rayOrigin, rayDir, 0.0f, t, n, NULL))
	    {	
			paths.totalRadiance[i] = n;
		}

		paths.mode[i] = ePathTerminate;
	}
}


struct GpuWaveFrontRenderer : public Renderer
{
	Color* output = NULL;
	
	GPUScene sceneGPU;
	
	Random rand;

	int tileWidth;
	int tileHeight;

	PathState paths;

	// map id to geometry struct
	std::map<int, MeshGeometry> gpuMeshes;

	GpuWaveFrontRenderer(const Scene* s)
	{
		// build GPU primitive and light lists
		std::vector<Primitive> primitives;		
		std::vector<Primitive> lights;

		for (int i=0; i < s->primitives.size(); ++i)
		{
			Primitive primitive = s->primitives[i];

			// if mesh primitive then copy to the GPU
			if (primitive.type == eMesh)
			{
				// see if we have already uploaded the mesh to the GPU
				if (gpuMeshes.find(primitive.mesh.id) == gpuMeshes.end())
				{
					MeshGeometry geo = CreateGPUMesh(primitive.mesh);
					gpuMeshes[geo.id] = geo;

					// replace CPU mesh with GPU copy
					primitive.mesh = geo;
				}
			}

			if (primitive.material.bump > 0.0f)
			{
				primitive.material.bumpMap = CreateGPUTexture(primitive.material.bumpMap);
			}
			
			// create explicit list of light primitives
			if (primitive.lightSamples)
			{
				lights.push_back(primitive);
			}

			primitives.push_back(primitive);
		}

		// convert scene BVH
		CreateVec4Texture((Vec4**)&(sceneGPU.bvh.nodes), (Vec4*)s->bvh.nodes, sizeof(BVHNode)*s->bvh.numNodes);
		sceneGPU.bvh.numNodes = s->bvh.numNodes;

		// upload to the GPU
		sceneGPU.numPrimitives = primitives.size();
		sceneGPU.numLights = lights.size();

		if (sceneGPU.numLights > 0)
		{
			hipMalloc(&sceneGPU.lights, sizeof(Primitive)*lights.size());
			hipMemcpy(sceneGPU.lights, &lights[0], sizeof(Primitive)*lights.size(), hipMemcpyHostToDevice);
		}

		if (sceneGPU.numPrimitives > 0)
		{
			hipMalloc(&sceneGPU.primitives, sizeof(Primitive)*primitives.size());
			hipMemcpy(sceneGPU.primitives, &primitives[0], sizeof(Primitive)*primitives.size(), hipMemcpyHostToDevice);
		}

		// copy sky and probe texture
		sceneGPU.sky = CreateGPUSky(s->sky);

		tileWidth = 1024;
		tileHeight = 1024;

		const int numPaths = tileWidth*tileHeight;

		// allocate paths
		//hipMalloc(&paths, sizeof(PathState)*numPaths);
		//hipMemset(paths, 0, sizeof(PathState)*numPaths);

		paths = AllocatePaths(numPaths);
	}

	virtual ~GpuWaveFrontRenderer()
	{
		hipFree(output);
		hipFree(sceneGPU.primitives);
		hipFree(sceneGPU.lights);
		
		FreePaths(paths);
	}
	
	void Init(int width, int height)
	{
		hipFree(output);
		hipMalloc(&output, sizeof(Color)*width*height);
		hipMemset(output, 0, sizeof(Color)*width*height);
	}

	void Render(const Camera& camera, const Options& options, Color* outputHost)
	{
		std::vector<Tile> tiles;

		const int tilesx = (options.width + tileWidth - 1)/tileWidth;
		const int tilesy = (options.height + tileHeight - 1)/tileHeight;

		for (int y=0; y < tilesy; ++y)
		{
			for (int x=0; x < tilesx; ++x)
			{
				Tile tile;
				tile.x = x*tileWidth;
				tile.y = y*tileHeight;

				tile.width = Min(tileWidth, options.width-tile.x);
				tile.height = Min(tileHeight, options.height-tile.y);

				tiles.push_back(tile);
			}
		}

		const int numPaths = tileWidth*tileHeight;

		// create a sampler for the camera
		CameraSampler sampler(
			Transform(camera.position, camera.rotation),
			camera.fov, 
			0.001f,
			1.0f,
			options.width,
			options.height);

		
		for (int tileIndex=0; tileIndex < tiles.size(); ++tileIndex)
		{
			Tile tile = tiles[tileIndex];

			// a tile consists of many thread blocks
			const int blockWidth = 16;
			const int blockHeight = 16;

			const int gridWidth = (tile.width + blockWidth - 1)/blockWidth;
			const int gridHeight = (tile.height + blockHeight - 1)/blockHeight;

			dim3 blockDim(blockWidth, blockHeight);
			dim3 gridDim(gridWidth, gridHeight);

			/*
			const int kNumThreadsPerBlock = 256;
			const int kNumBlocks = (numPaths + kNumThreadsPerBlock - 1)/kNumThreadsPerBlock;
			
			dim3 gridDim(kNumBlocks);
			dim3 blockDim(kNumThreadsPerBlock);
			*/

			GeneratePaths<<<gridDim, blockDim>>>(camera, sampler, tile, rand.Rand(), paths, numPaths);
	
			if (options.mode == eNormals)
			{
				VisualizeNormals<<<gridDim, blockDim>>>(sceneGPU, paths, numPaths);
			}
			else
			{
				for (int i=0; i < options.maxDepth; ++i)
				{
					AdvancePaths<<<gridDim, blockDim>>>(sceneGPU, paths, numPaths);
					SampleLights<<<gridDim, blockDim>>>(sceneGPU, paths, numPaths);
					//SampleProbes();
					SampleBsdfs<<<gridDim, blockDim>>>(paths, numPaths);
				}
			}
			

			TerminatePaths<<<gridDim, blockDim>>>(output, options, paths, numPaths);
		}

		// copy back to output
		hipMemcpy(outputHost, output, sizeof(Color)*options.width*options.height, hipMemcpyDeviceToHost);
	}
};


Renderer* CreateGpuWavefrontRenderer(const Scene* s)
{
	return new GpuWaveFrontRenderer(s);
}
